#ifndef APPEL_GPU_DISABLED

#include <gpu/manager.h>
#include <gpu/pragma.h>
#include <gpu/types/triangle.h>
#include <gpu/kernel.h>
#include <gpu/types/ray.h>

Manager::Manager(unsigned int maxTriangles) {
    this->maxTriangles = maxTriangles;

    size_t size = maxTriangles * sizeof(GTriangle);
    CUDA_STATUS(hipMalloc((void**)&cache, size));
    CUDA_STATUS(hipMallocManaged((void**)&buffer, sizeof(int)));

    CUDA_STATUS(hipMallocManaged((void**)&dvc_ray, sizeof(GRay)));

    CUDA_STATUS(hipMallocManaged((void**)&dvc_N, sizeof(int)));
    CUDA_STATUS(hipDeviceSynchronize());

    dvc_N[0] = (int)maxTriangles;

    for(int i=0;i<(int)maxTriangles;i++) {
        GTriangle t;
        t.host_id = -1;
        updateCache<<<1,1>>>(i, t, cache);
        free_pos.push(i);
    }

    CUDA_STATUS(hipDeviceSynchronize());
}

Manager::~Manager() {
    CUDA_STATUS(hipFree(buffer));
    CUDA_STATUS(hipFree(cache));
    CUDA_STATUS(hipFree(dvc_N));
    CUDA_STATUS(hipFree(dvc_ray));
}

#endif
