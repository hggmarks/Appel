#ifndef APPEL_GPU_DISABLED

#include <gpu/manager.h>
#include <gpu/pragma.h>
#include <gpu/types/triangle.h>
#include <gpu/kernel.h>

Manager::Manager(unsigned int maxTriangles) {
    this->maxTriangles = maxTriangles;

    size_t size = maxTriangles * sizeof(GTriangle);
    CUDA_STATUS(hipMalloc((void**)&cache, size));
    CUDA_STATUS(hipMalloc((void**)&buffer, sizeof(int)));
    CUDA_STATUS(hipDeviceSynchronize());

    for(int i=0;i<(int)maxTriangles;i++) {
        GTriangle t;
        t.host_id = -1;
        updateCache<<<1,1>>>(i, t, cache);
        free_pos.push(i);
    }

    CUDA_STATUS(hipDeviceSynchronize());
}

Manager::~Manager() {
    CUDA_STATUS(hipFree(buffer));
    CUDA_STATUS(hipFree(cache));
}

#endif
