#include "hip/hip_runtime.h"
#include <gpu/helper.h>

__device__ GPoint normalize(GPoint p) {
    float r = sqrt(p.x * p.x + p.y * p.y + p.z * p.z);
    p.x /= r;
    p.y /= r;
    p.z /= r;
    
    return p;
}