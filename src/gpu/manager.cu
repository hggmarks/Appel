#include <gpu/manager.h>
#include <gpu/types/ray.h>
#include <gpu/kernel.h>

Manager::Manager() {
    calls_counter = 0;
    cache_limit = 100000;
    tableFrequency = std::set<std::pair<int, int>>();
    hostToDeviceID = std::map<int, int>();

    hipMalloc(&cache, cache_limit * sizeof(GTriangle));

    for(int i=0;i<cache_limit;i++) {
        GTriangle gt;
        gt.host_id = -1;
        hipMemcpy(&cache[i], &gt, 1, hipMemcpyHostToDevice);
    }

    int threadsperblock = 1024;
    int blockspergrid = (cache_limit + threadsperblock - 1) / threadsperblock;

    int *block_idx = (int*)malloc(sizeof(int) * blockspergrid);
    float *block_dist = (float*)malloc(sizeof(float) * blockspergrid);

    int *dvc_block_idx;
    float *dvc_block_dist;

    hipMalloc(&dvc_block_idx, blockspergrid * sizeof(int));
    hipMalloc(&dvc_block_dist, blockspergrid * sizeof(float));
}

Manager::~Manager() {
    hipFree(cache);
    hipFree(dvc_block_idx);
    hipFree(dvc_block_dist);

    free(block_idx);
    free(block_dist);
}

void Manager::transfer(int host_id, const Triangle &triangle) {
    if(isOnCache(host_id)) return;

    int device_id = getFreeDeviceId();
    GTriangle gt;
    gt.p0x = triangle.getVertex(0).x;
    gt.p0y = triangle.getVertex(0).y;
    gt.p0z = triangle.getVertex(0).z;

    gt.p1x = triangle.getVertex(1).x;
    gt.p1y = triangle.getVertex(1).y;
    gt.p1z = triangle.getVertex(1).z;

    gt.p2x = triangle.getVertex(2).x;
    gt.p2y = triangle.getVertex(2).y;
    gt.p2z = triangle.getVertex(2).z;

    gt.host_id = host_id;

    lazy.push_back(std::make_pair(device_id, gt));
}

bool Manager::isOnCache(int host_id) {
    auto it = tableFrequency.lower_bound(std::make_pair(host_id, -1));
    if(it == tableFrequency.end() || (*it).first != host_id) return false;

    tableFrequency.erase(it);
    tableFrequency.insert(std::make_pair(host_id, calls_counter));

    hostToDeviceID[host_id] = calls_counter;

    return true;
}

int Manager::getFreeDeviceId() {
    if((int)tableFrequency.size() < cache_limit) return tableFrequency.size();

    auto it = tableFrequency.begin();
    int host_id = (*it).first;
    int device_id = hostToDeviceID.at(host_id);
    
    hostToDeviceID.erase(host_id);
    tableFrequency.erase(it);

    return device_id;
}

void Manager::pendingTransfer() {
    hipStream_t streams[lazy.size()];

    for(int i=0;i<(int)lazy.size();i++) {
        int dvc_id = lazy[i].first;
        GTriangle gt = lazy[i].second;

        hipMemcpyAsync(&cache[dvc_id], &gt, 1, hipMemcpyHostToDevice, streams[i]);
    }

    for(int i=0;i<(int)lazy.size();i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    lazy.clear();
}

int Manager::run(const Ray &ray) {
    pendingTransfer();
    GRay gr;
    gr.lx = ray.location.x;
    gr.ly = ray.location.y;
    gr.lz = ray.location.z;

    gr.dx = ray.direction.x;
    gr.dy = ray.direction.y;
    gr.dz = ray.direction.z;

    GRay *dvc_gr;
    hipMalloc(&dvc_gr, sizeof(GRay));
    hipMemcpy(&dvc_gr, &gr, sizeof(GRay), hipMemcpyHostToDevice);

    castRay<<<blockspergrid, threadsperblock>>>(cache, dvc_gr, dvc_block_dist, dvc_block_idx, cache_limit);
    hipDeviceSynchronize();

    hipMemcpy(dvc_block_dist, block_dist, sizeof(float) * blockspergrid, hipMemcpyDeviceToHost);
    hipMemcpy(dvc_block_idx, block_idx, sizeof(int) * blockspergrid, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    float minT = MAXFLOAT;
    int idx = -1;

    for(int i=0;i<blockspergrid;i++){
        if(block_dist[i] > 0.0 && block_dist[i] < minT) {
            minT = block_dist[i];
            idx = block_idx[i];
        }
    }

    hipFree(dvc_gr);

    return idx;
}
